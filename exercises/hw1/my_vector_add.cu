
#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheckErrors(msg) \
  do { \
    hipError_t __err = hipGetLastError(); \
    if (__err != hipSuccess) { \
      fprintf(stderr, "Fatal error: %s (%s at %s:%d)\n", \
          msg, hipGetErrorString(__err), \
          __FILE__, __LINE__); \
      fprintf(stderr, "*** FAILED - ABORTING\n"); \
      exit(1);\
    }\
  } while (0) 

__global__ void vector_add(float* a, float* b, float* c, int N){
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  if (i < N) c[i] = b[i] + a[i]; 
}


int main() {
  int n = 1000;
  int threadPerBlock = 256;
  size_t size = n * sizeof(float);
  
  float* a_h = (float*)malloc(size);
  float* b_h = (float*)malloc(size);
  float* c_h = (float*)malloc(size);

  for (int i=0; i<n; i++) {
    a_h[i] = rand()/(float)RAND_MAX;
    b_h[i] = rand()/(float)RAND_MAX;
    c_h[i] = 0;
  }

  float* a_d;
  hipMalloc(&a_d, size);
  float* b_d;
  hipMalloc(&b_d, size);
  float* c_d;
  hipMalloc(&c_d, size);

  cudaCheckErrors("hipMalloc failed");
  // Send data to device
  hipMemcpy(a_d, a_h, size, hipMemcpyHostToDevice);
  hipMemcpy(b_d, b_h, size, hipMemcpyHostToDevice);

  cudaCheckErrors("hipMemcpy failed");
  // launch the kernel
  vector_add<<<(n+threadPerBlock-1)/threadPerBlock, threadPerBlock>>>(a_d, b_d, c_d, n);

  cudaCheckErrors("kernel launch failed");

  // Get data from device
  hipMemcpy(c_h, c_d, size, hipMemcpyDeviceToHost);

  cudaCheckErrors("kernel or cudaMem failed");

  printf("A[0]:%f\n", a_h[0]);
  printf("B[0]:%f\n", b_h[0]);
  printf("C[0]:%f\n", c_h[0]);
  
  free(a_h);
  free(b_h);
  free(c_h);
  // free
  hipFree(a_d);
  hipFree(b_d);
  hipFree(c_d);

}
